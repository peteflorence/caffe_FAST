#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

// convert map to Blob because CUDA code cannot access map STL
void GetMapData(const map<int, vector<int> >& group_map,
    Blob<int>* group_map_range, Blob<int>* group_map_index = NULL) {
  // get the start and end index of all groups
  group_map_range->Reshape(1, 1, 1, group_map.size()+1);
  int* group_map_range_data = group_map_range->mutable_cpu_data();
  int total_count = 0;
  int count = 0;
  for (map<int, vector<int> >::const_iterator it = group_map.begin();
      it != group_map.end(); ++it) {
    group_map_range_data[count] = total_count;
    total_count += it->second.size();
    ++count;
  }
  group_map_range_data[count] = total_count;
  // get the group_map_index if necessary
  if (group_map_index != NULL) {
    group_map_index->Reshape(1, 1, 1, total_count);
    int* group_map_index_data = group_map_index->mutable_cpu_data();
    count = 0;
    for (map<int, vector<int> >::const_iterator it = group_map.begin();
        it != group_map.end(); ++it) {
      for (int s = 0; s < it->second.size(); ++s) {
        group_map_index_data[count] = it->second[s];
        ++count;
      }
    }
  }
}

template <typename Dtype>
__global__ void ComputeGroupMean(const int nthreads, const Dtype* data,
    const int channels, const int height, const int width, const int num_groups,
    const int* group_map_range, const int* group_map_index,
    const int group_channels, Dtype* group_mean_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int group_id = index % num_groups;
    int c = (index / num_groups) % channels;
    data += c * height * width;
    int start_idx = group_map_range[group_id];
    int end_idx = group_map_range[group_id + 1];
    Dtype sumval = 0;
    for (int i = start_idx; i < end_idx; ++i) {
      sumval += data[group_map_index[i]];
    }
    group_mean_data[index] = sumval / (end_idx - start_idx) / group_channels;
  }
}

template <typename Dtype>
__global__ void GroupUnPoolForward(const int nthreads, const Dtype* group_mean_data,
    const int channels, const int height, const int width, const Dtype* group_data,
    const int num_groups, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int group_id = (int)group_data[h * width + w];
    top_data[index] += group_mean_data[c * num_groups + group_id];
  }
}

template <typename Dtype>
void GroupingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  // use the reordered internal group data
  const Dtype* group_data = group_blob_.gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  caffe_gpu_set(top[0]->count(), Dtype(0), top_data);
  const int dim = top[0]->count() / num_;
  // NOLINT_NEXT_LINE(whitespace/operators)
  CHECK_EQ(bottom.size(), 2);
  for (int n = 0; n < num_; ++n) {
    const vector<map<int, vector<int> > >& group_maps = group_maps_vec_[n];
    for (int gc = 0; gc < group_channels_; ++gc) {
      const map<int, vector<int> >& group_map = group_maps[gc];
      const int num_groups = group_map.size();
      // cuda function cannot call STL function, convert it to Blob data
      GetMapData(group_map, &group_map_range_, &group_map_index_);
      const int* group_map_range = group_map_range_.gpu_data();
      const int* group_map_index = group_map_index_.gpu_data();
      // prepare group_mean_
      group_mean_.Reshape(1, channels_, 1, num_groups);
      Dtype* group_mean_data = group_mean_.mutable_gpu_data();
      int group_count = group_mean_.count();
      // compute group_mean_data
      ComputeGroupMean<Dtype><<<CAFFE_GET_BLOCKS(group_count), CAFFE_CUDA_NUM_THREADS>>>(
          group_count, bottom_data, channels_, height_, width_, num_groups,
          group_map_range, group_map_index, group_channels_, group_mean_data);
      // spread group_mean_data to top_data
      GroupUnPoolForward<Dtype><<<CAFFE_GET_BLOCKS(dim), CAFFE_CUDA_NUM_THREADS>>>(
          dim, group_mean_data, channels_, height_, width_,
          group_data, num_groups, top_data);
      group_data += bottom[1]->offset(0, 1);
    }
    bottom_data += bottom[0]->offset(1);
    top_data += top[0]->offset(1);
  }
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void GroupUnPoolBackward(const int nthreads, const Dtype* group_mean_diff,
    const int channels, const int height, const int width, const Dtype* group_data,
    const int num_groups, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int group_id = (int)group_data[h * width + w];
    bottom_diff[index] += group_mean_diff[c * num_groups + group_id];
  }
}

template <typename Dtype>
void GroupingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  // use the reordered internal group data
  const Dtype* group_data = group_blob_.gpu_data();
  const int dim = bottom[0]->count() / top[0]->num();
  caffe_gpu_set(bottom[0]->count(), Dtype(0.), bottom_diff);
  // NOLINT_NEXT_LINE(whitespace/operators)
  CHECK_EQ(bottom.size(), 2);
  for (int n = 0; n < num_; ++n) {
    const vector<map<int, vector<int> > >& group_maps = group_maps_vec_[n];
    for (int gc = 0; gc < group_channels_; ++gc) {
      const map<int, vector<int> >& group_map = group_maps[gc];
      const int num_groups = group_map.size();
      // cuda function cannot call STL function, convert it to Blob data
      GetMapData(group_map, &group_map_range_, &group_map_index_);
      const int* group_map_range = group_map_range_.gpu_data();
      const int* group_map_index = group_map_index_.gpu_data();
      // prepare group_mean_
      group_mean_.Reshape(1, channels_, 1, num_groups);
      Dtype* group_mean_diff = group_mean_.mutable_gpu_diff();
      int group_count = group_mean_.count();
      // compute group_mean_diff
      ComputeGroupMean<Dtype><<<CAFFE_GET_BLOCKS(group_count), CAFFE_CUDA_NUM_THREADS>>>(
          group_count, top_diff, channels_, height_, width_, num_groups,
          group_map_range, group_map_index, group_channels_, group_mean_diff);
      // spread group_mean_diff to bottom_diff
      GroupUnPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(dim), CAFFE_CUDA_NUM_THREADS>>>(
          dim, group_mean_diff, channels_, height_, width_, group_data,
          num_groups, bottom_diff);
      group_data += bottom[1]->offset(0, 1);
    }
    bottom_diff += bottom[0]->offset(1);
    top_diff += top[0]->offset(1);
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(GroupingLayer);


}  // namespace caffe
