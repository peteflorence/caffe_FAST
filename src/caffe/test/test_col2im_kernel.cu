#include "hip/hip_runtime.h"
#include <cstring>
#include <vector>

#include "gtest/gtest.h"

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/vision_layers.hpp"

#include "caffe/test/test_caffe_main.hpp"

namespace caffe {

// Forward declare kernel functions
template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels,
    const int patch_h, const int patch_w, const int pad_h, const int pad_w,
    const int stride_h, const int stride_w, const int filter_stride_h,
    const int filter_stride_w, const int height_col, const int width_col,
    Dtype* data_im);

extern hipDeviceProp_t CAFFE_TEST_CUDA_PROP;

template <typename Dtype>
class Col2ImKernelTest : public ::testing::Test {
 protected:
  Col2ImKernelTest()
        // big so launches > 1024 threads
      : blob_bottom_(new Blob<Dtype>(5, 500, 10, 10)),
        blob_top_(new Blob<Dtype>()),
        blob_top_cpu_(new Blob<Dtype>()) {
    FillerParameter filler_param;
    GaussianFiller<Dtype> filler(filler_param);
    filler.Fill(this->blob_bottom_);

    height_col_ = blob_bottom_->height();
    width_col_ = blob_bottom_->width();
    channels_col_ = blob_bottom_->channels();
    pad_ = 1;
    stride_ = 1;
    filter_stride_ = 2;
    kernel_size_ = 2;
    channels_ = channels_col_ / kernel_size_ / kernel_size_;
    const int kernel_size_eff = kernel_size_
      + (kernel_size_ - 1) * (filter_stride_ - 1);
    height_ = (height_col_ - 1) * stride_ + kernel_size_eff - 2 * pad_;
    width_ = (width_col_ - 1) * stride_ + kernel_size_eff - 2 * pad_;
  }

  virtual ~Col2ImKernelTest() {
      delete blob_bottom_;
      delete blob_top_;
      delete blob_top_cpu_;
  }

  Blob<Dtype>* const blob_bottom_;
  Blob<Dtype>* const blob_top_;
  Blob<Dtype>* const blob_top_cpu_;
  int height_;
  int width_;
  int channels_;
  int pad_;
  int stride_;
  int filter_stride_;
  int kernel_size_;
  int height_col_;
  int width_col_;
  int channels_col_;
};

TYPED_TEST_CASE(Col2ImKernelTest, TestDtypes);

TYPED_TEST(Col2ImKernelTest, TestGPU) {
  Caffe::set_mode(Caffe::GPU);

  // Reshape the blobs to correct size for Col2Im output
  this->blob_top_->Reshape(this->blob_bottom_->num(),
          this->channels_, this->height_, this->width_);

  this->blob_top_cpu_->Reshape(this->blob_bottom_->num(),
          this->channels_, this->height_, this->width_);

  const TypeParam* bottom_data = this->blob_bottom_->gpu_data();
  TypeParam* top_data = this->blob_top_->mutable_gpu_data();
  TypeParam* cpu_data = this->blob_top_cpu_->mutable_cpu_data();

  // CPU Version
  for (int n = 0; n < this->blob_bottom_->num(); ++n) {
    col2im_cpu(this->blob_bottom_->cpu_data() + this->blob_bottom_->offset(n),
      this->channels_, this->height_, this->width_,
      this->kernel_size_, this->kernel_size_, this->pad_, this->pad_,
      this->stride_, this->stride_, this->filter_stride_, this->filter_stride_,
      cpu_data + this->blob_top_cpu_->offset(n));
  }

  // GPU version
  int num_kernels = this->channels_ * this->height_ * this->width_;
  int default_grid_dim = CAFFE_GET_BLOCKS(num_kernels);

  // Launch with different grid sizes
  for (int grid_div = 2; grid_div <= 8; grid_div++) {
    for (int n = 0; n < this->blob_bottom_->num(); ++n) {
      int grid_dim = default_grid_dim/grid_div;
      // NOLINT_NEXT_LINE(whitespace/operators)
      col2im_gpu_kernel<TypeParam><<<grid_dim, CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, bottom_data + this->blob_bottom_->offset(n),
        this->height_, this->width_, this->channels_, this->kernel_size_,
        this->kernel_size_, this->pad_, this->pad_, this->stride_,
        this->stride_, this->filter_stride_, this->filter_stride_,
        this->height_col_, this->width_col_,
        top_data + this->blob_top_->offset(n));
      CUDA_POST_KERNEL_CHECK;
    }

    // Compare results against CPU version
    for (int i = 0; i < this->blob_top_->count(); ++i) {
      TypeParam cpuval = cpu_data[i];
      TypeParam gpuval = this->blob_top_->cpu_data()[i];
      EXPECT_NEAR(cpuval, gpuval, 1e-5);
    }
  }
}

}  // namespace caffe
